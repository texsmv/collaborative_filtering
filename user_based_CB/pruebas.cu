#include "hip/hip_runtime.h"
#include "scripts.h"
#include <iostream>
#include "structures.h"
#include "distances.h"
#include "cud_sparse_oper.h"
#include "recomender.h"

using namespace std;


int main(int argc, char const *argv[]) {
  int n_ratings, n_users;
  n_ratings = 27753444;
  n_users = 283228;
  // n_of_users("../databases/ml-latest/ratings.csv", n_ratings, n_users, true);
  // cout<<n_ratings<<" "<<n_users<<endl;

  float* values;
  int *row_ind, * col_ind;
  int * ind_users, *row_size;

  float* d_values;
  int *d_row_ind, * d_col_ind;
  int * d_ind_users, * d_row_size;

  d_values = cuda_array<float>(n_ratings);
  d_row_ind = cuda_array<int>(n_ratings);
  d_col_ind = cuda_array<int>(n_ratings);
  d_ind_users = cuda_array<int>(n_users);
  d_row_size = cuda_array<int>(n_users);


  read_ML("../databases/ml-latest/ratings.csv", n_ratings, n_users, true, values, row_ind, col_ind, ind_users, row_size);

  cuda_H2D<float>(values, d_values, n_ratings);
  cuda_H2D<int>(row_ind, d_row_ind, n_ratings);
  cuda_H2D<int>(col_ind, d_col_ind, n_ratings);
  cuda_H2D<int>(ind_users, d_ind_users, n_users);
  cuda_H2D<int>(row_size, d_row_size, n_users);


  // float* r1 = float_pointer(values, ind_users, 3);
  // int* c1 = int_pointer(col_ind, ind_users, 3);
  //
  // float* r2 = float_pointer(values, ind_users, 1);
  // int* c2 = int_pointer(col_ind, ind_users, 1);
  //
  // cout<<euclidean(r1, c1, row_size[3], r2, c2, row_size[1])<<endl;




  vector<int> ids_movies;
  vector<float> movies_ratings;

  k_recomendations(ids_movies, movies_ratings, d_values, d_row_ind, d_col_ind, d_ind_users, d_row_size, values, row_ind, col_ind, ind_users, row_size, n_ratings, n_users, EUCLIDEAN, 3242, 10);
  //
  // float* distances = new float[n_users];
  // float* d_distances;
  // d_distances = cuda_array<float>(n_users);
  //
  //
  //
  //
  // int block_size = 256;
  // dim3 block =  dim3(block_size, 1, 1);
  // dim3 grid =  dim3(ceil(n_users / block_size), 1);
  //
  //
  // one2all_pearson<<<grid, block>>>(d_values, d_row_ind, d_col_ind, d_ind_users, d_row_size, d_distances, 3, n_users);
  // CHECK(hipDeviceSynchronize());
  // cuda_D2H<float>(d_distances, distances, n_users);
  // CHECK(hipDeviceSynchronize());
  //
  // cout<<"--------"<<endl;
  // for (size_t i = 0; i < 10; i++) {
  //   cout<<distances[i]<<endl;
  // }
  //
  // float* distances2 = new float[n_users];
  //
  //
  //
  // float v;
  // for (size_t i = 3; i < 4; i++) {
  //   float* r1 = float_pointer(values, ind_users, i);
  //   int* c1 = int_pointer(col_ind, ind_users, i);
  //   // if(i % 1000 == 0)
  //   // cout<<i<<endl;
  //   for (size_t j = 0; j < 10; j++) {
  //     float* r2 = float_pointer(values, ind_users, j);
  //     int* c2 = int_pointer(col_ind, ind_users, j);
  //     v = pearson(r1, c1, row_size[i], r2, c2, row_size[j]);
  //     distances2[j] = v;
  //     // if(v != 0)
  //     //   cout<<v<<endl;
  //
  //   }
  // }
  //
  // cout<<"--------"<<endl;
  // for (size_t i = 0; i < 10; i++) {
  //   cout<<distances[i]<<endl;
  // }

  return 0;
}
