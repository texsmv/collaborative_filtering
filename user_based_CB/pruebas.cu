#include "hip/hip_runtime.h"
#include "scripts.h"
#include <iostream>
#include "structures.h"
#include "distances.h"
#include "cud_sparse_oper.h"
#include "recomender.h"


using namespace std;


int main(int argc, char const *argv[]) {




  int n_ratings, n_users;
  int n_ratings_20, n_users_20, n_ratings_27, n_users_27;

  n_ratings_27 = 27753444;
  n_users_27 = 283228;

  n_ratings_20 = 20000263;
  n_users_20 = 138493;

  n_ratings = n_ratings_20;
  n_users = n_users_20;
  // n_ratings
  // n_of_users("../databases/ml-20m/ratings.csv", n_ratings, n_users, true);
  // cout<<n_ratings<<" "<<n_users<<endl;

  float* values;
  int *row_ind, * col_ind;
  int * ind_users, *row_size;

  float* d_values;
  int *d_row_ind, * d_col_ind;
  int * d_ind_users, * d_row_size;

  d_values = cuda_array<float>(n_ratings);
  d_row_ind = cuda_array<int>(n_ratings);
  d_col_ind = cuda_array<int>(n_ratings);
  d_ind_users = cuda_array<int>(n_users);
  d_row_size = cuda_array<int>(n_users);


  read_ML("../databases/ml-20m/ratings.csv", n_ratings, n_users, true, values, row_ind, col_ind, ind_users, row_size);

  cuda_H2D<float>(values, d_values, n_ratings);
  cuda_H2D<int>(row_ind, d_row_ind, n_ratings);
  cuda_H2D<int>(col_ind, d_col_ind, n_ratings);
  cuda_H2D<int>(ind_users, d_ind_users, n_users);
  cuda_H2D<int>(row_size, d_row_size, n_users);


  // float* r1 = float_pointer(values, ind_users, 283227);
  // int* c1 = int_pointer(col_ind, ind_users, 283227);
  // cout<<"tam: "<<row_size[283227]<<endl;
  // for (size_t i = 0 ; i < row_size[283227]; i++) {
  //   cout<<i<<" movie: "<<c1[i]<<" - "<<r1[i]<<endl;
  // }



  // float* r1 = float_pointer(values, ind_users, 3);
  // int* c1 = int_pointer(col_ind, ind_users, 3);
  //
  // float* r2 = float_pointer(values, ind_users, 1);
  // int* c2 = int_pointer(col_ind, ind_users, 1);
  //
  // cout<<euclidean(r1, c1, row_size[3], r2, c2, row_size[1])<<endl;


  int pos;
  int id_movie;
  while (true) {
    cin>>pos;
    cin>>id_movie;
    vector<int> ids_movies;
    vector<float> movies_ratings;
    reloj r;
    r.start();
    // k_proyection(ids_movies, movies_ratings, d_values, d_row_ind, d_col_ind, d_ind_users, d_row_size, values, row_ind, col_ind, ind_users, row_size, n_ratings, n_users,COSINE, pos, id_movie,n_users);
    k_recomendations(ids_movies, movies_ratings, d_values, d_row_ind, d_col_ind, d_ind_users, d_row_size, values, row_ind, col_ind, ind_users, row_size, n_ratings, n_users, COSINE, pos, 100);
    r.stop();
    cout<<r.time()<<"ms"<<endl;
    for (size_t i = 0; i < ids_movies.size(); i++) {
      cout<<ids_movies[i]<<" "<<movies_ratings[i]<<endl;
    }
  }

  // float* distances = new float[n_users];
  // float* d_distances;
  // d_distances = cuda_array<float>(n_users);
  //
  //
  //
  //
  // int block_size = 256;
  // dim3 block =  dim3(block_size, 1, 1);
  // dim3 grid =  dim3(ceil(n_users / block_size), 1);
  //
  //
  // one2all_pearson<<<grid, block>>>(d_values, d_row_ind, d_col_ind, d_ind_users, d_row_size, d_distances, 3, n_users);
  // CHECK(hipDeviceSynchronize());
  // cuda_D2H<float>(d_distances, distances, n_users);
  // CHECK(hipDeviceSynchronize());
  //
  // cout<<"--------"<<endl;
  // for (size_t i = 0; i < 10; i++) {
  //   cout<<distances[i]<<endl;
  // }
  //
  // float* distances2 = new float[n_users];
  //
  //
  //
  // float v;
  // for (size_t i = 3; i < 4; i++) {
  //   cout<<i<<endl;
  //   float* r1 = float_pointer(values, ind_users, i);
  //   int* c1 = int_pointer(col_ind, ind_users, i);
  //   // if(i % 1000 == 0)
  //   // cout<<i<<endl;
  //   for (size_t k = 0; k < row_size[i]; k++) {
  //     cout<<r1[k]<<" -- "<<c1[k]<<endl;
  //   }
  //   cout<<"------------------"<<endl;
  //   cout<<"------------------"<<endl;
  //   cout<<"------------------"<<endl;
  //   for (size_t j = 283227; j < n_users; j++) {
  //     cout<<"j: "<<j<<endl;
  //     float* r2 = float_pointer(values, ind_users, j);
  //     int* c2 = int_pointer(col_ind, ind_users, j);
  //     if(j == 283227){
  //         for (size_t k = 0; k < row_size[j]; k++) {
  //           cout<<r2[k]<<" -- "<<c2[k]<<endl;
  //         }
  //     }
  //
  //     cout<<"-"<<endl;
  //     v = cosine(r1, c1, row_size[i], r2, c2, row_size[j]);
  //     cout<<"-"<<endl;
  //     if(std::isnan(v)){
  //       cout<<"nan"<<endl;
  //     }
  //     distances2[j] = v;
  //     cout<<"-"<<endl;
  //     // if(v != 0)
  //     //   cout<<v<<endl;
  //
  //   }
  // }

  // cout<<"--------"<<endl;
  // for (size_t i = 0; i < 200; i++) {
  //   if (distances2[i] != 0)
  //     cout<<distances2[i]<<endl;
  // }
  //
  return 0;
}
