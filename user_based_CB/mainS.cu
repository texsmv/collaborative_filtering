#include "hip/hip_runtime.h"
#include "sparse_operations.h"
#include <fstream>

#include <cstdlib>
#include <ctime>



int main(int argc, char const *argv[]) {



  float* values;
  int *row_ind, * col_ind;
  int * ind_users, *row_size;

  float* d_values;
  int *d_row_ind, * d_col_ind;
  int * d_ind_users, * d_row_size;

  float* distances = new float[n_users];
  float* d_distances;

  clock_t start2 = clock();
  // read_movieLens("ml-20millones/ratings.csv", values, row_ind, col_ind, ind_users, row_size);
  // read_movieLens("ml-latest/ratings.csv", values, row_ind, col_ind, ind_users, row_size);
  read_movieLens("ml-20m/ratings.csv", values, row_ind, col_ind, ind_users, row_size);
  clock_t stop2 = clock();
  double elapsed2 = (double)(stop2 - start2) * 1000.0 / CLOCKS_PER_SEC;
  cout<<"Tiempo de cargar bd:  "<<elapsed2<<"ms"<<endl;

  d_values = cuda_array<float>(n_ratings);
  d_row_ind = cuda_array<int>(n_ratings);
  d_col_ind = cuda_array<int>(n_ratings);
  d_ind_users = cuda_array<int>(n_users);
  d_row_size = cuda_array<int>(n_users);

  d_distances = cuda_array<float>(n_users);

  cuda_H2D<float>(values, d_values, n_ratings);
  cuda_H2D<int>(row_ind, d_row_ind, n_ratings);
  cuda_H2D<int>(col_ind, d_col_ind, n_ratings);
  cuda_H2D<int>(ind_users, d_ind_users, n_users);
  cuda_H2D<int>(row_size, d_row_size, n_users);

  int block_size = 256;
  dim3 block =  dim3(block_size, 1, 1);
  dim3 grid =  dim3(ceil(n_users / block_size), 1);


  clock_t start3 = clock();
  one2all<<<grid, block>>>(d_values, d_row_ind, d_col_ind, d_ind_users, d_row_size, d_distances, 3);
  clock_t stop3 = clock();
  double elapsed3 = (double)(stop3 - start3) * 1000.0 / CLOCKS_PER_SEC;
  cout<<"Tiempo uno a todos gpu:  "<<elapsed3<<"ms"<<endl;




  CHECK(hipDeviceSynchronize());
  cuda_D2H<float>(d_distances, distances, n_users);






  // d_row_ind = <int> cuda_array(n_ratings);

  // for (size_t i = 0; i < 10; i++) {
  //   cout<<values[i]<<" --> "<<row_ind[i]<<" "<<col_ind[i]<<endl;
  // }
  //
  // for (size_t i = 0; i < 4; i++) {
  //   cout<<"Ind: "<<ind_users[i]<<" stride: "<<row_size[i]<<endl;
  // }
  // cout<<"Ind: "<<ind_users[n_users - 1]<<" stride: "<<row_size[n_users - 1]<<endl;
  //
  //
  // float* r = float_pointer(values, ind_users, 1);
  //
  // for (size_t i = 0; i < row_size[1]; i++) {
  //   cout<<r[i]<<" - ";
  // }
  // cout<<endl;
  //


  float* distances2 = new float[n_users];

  clock_t start4 = clock();


  float v;
  for (size_t i = 3; i < 4; i++) {
    float* r1 = float_pointer(values, ind_users, i);
    int* c1 = int_pointer(col_ind, ind_users, i);
    // if(i % 1000 == 0)
    // cout<<i<<endl;
    for (size_t j = 0; j < n_users; j++) {
      float* r2 = float_pointer(values, ind_users, j);
      int* c2 = int_pointer(col_ind, ind_users, j);
      v = euclidean(r1, c1, row_size[i], r2, c2, row_size[j]);
      distances2[j] = v;
      // if(v != 0)
      //   cout<<v<<endl;

    }
  }

  clock_t stop4 = clock();
  double elapsed4 = (double)(stop4 - start4) * 1000.0 / CLOCKS_PER_SEC;
  cout<<"Tiempo uno a todos cpu:  "<<elapsed4<<"ms"<<endl;


  // for (size_t i = 0; i < 100; i++) {
  //   cout<<distances2[i]<< " - "<<distances[i]<<endl;
  // }



  // float *dists;
  // int *ids;
  // clock_t start4 = clock();
  // n_knns(dists, ids, 0, 1, d_values, d_row_ind, d_col_ind, d_ind_users, d_row_size);
  // clock_t stop4 = clock();
  // double elapsed4 = (double)(stop4 - start4) * 1000.0 / CLOCKS_PER_SEC;
  // cout<<"Tiempo proyeccion:  "<<elapsed4<<"ms"<<endl;



  // knn(distances, ids, dists, 0, n_users);
  // cout<<"Mas cercanos"<<endl;
  // for (size_t i = 0; i < n_knn * 4 ; i++) {
  //   if(dists[i] < numeric_limits<float>::infinity())
  //     cout<<ids[i]<<" -> "<<dists[i]<<endl;
  // }

  // std::ofstream ofile("foobar.bin", std::ios::binary);
  // ofile.write((char*)dists, sizeof(float) * n_knn * 4);
  //
  // std::ofstream ofile2("foobar2.bin", std::ios::binary);
  // ofile2.write((char*)ids, sizeof(int) * n_knn * 4);


  start3 = clock();
  vector<int> recomendados = top_k_movies(d_values, d_row_ind, d_col_ind, d_ind_users, d_row_size, values, row_ind, col_ind, ind_users, row_size, 3, 100);
  for (size_t i = 0; i < recomendados.size(); i++) {
    // cout<<recomendados[i]<<" ";
  }
  cout<<endl;
  stop3 = clock();
  elapsed3 = (double)(stop3 - start3) * 1000.0 / CLOCKS_PER_SEC;
  cout<<"Tiempo top k:  "<<elapsed3<<"ms"<<endl;


  start3 = clock();
  float proyeccion = proyeccion_movies(d_values, d_row_ind, d_col_ind, d_ind_users, d_row_size, values, row_ind, col_ind, ind_users, row_size, 3, 10);
  stop3 = clock();
  elapsed3 = (double)(stop3 - start3) * 1000.0 / CLOCKS_PER_SEC;
  cout<<"Valor proyectado:  "<<proyeccion<<endl;
  cout<<"Tiempo proyeccion:  "<<elapsed3<<"ms"<<endl;



  int id;
  int id_pelicula;
  int k;
  while (true) {
    int opcion;
    cout<<"Id del usuario:"<<endl;
    // id =0;
    cin>>id;
    cout<<endl;
    // t_id_user id = db[0].f;
    // t_ratings r = *(db[0].s);
    // tree.get_sm(r, sm, rm);
    cout<<"1. para top K mas cercanos: "<<endl;
    cout<<"2. para prediccion dado k vecinos: "<<endl;

    cin>>opcion;
    // cout<<"Ingrese k : "<<endl;
    // cin>>k;

    if(opcion == 1){

      clock_t start4 = clock();
      vector<int> recomendados = top_k_movies(d_values, d_row_ind, d_col_ind, d_ind_users, d_row_size, values, row_ind, col_ind, ind_users, row_size, id, 100);
      clock_t stop4 = clock();
      double elapsed4 = (double)(stop4 - start4) * 1000.0 / CLOCKS_PER_SEC;
      cout<<"Tiempo :  "<<elapsed4<<"ms"<<endl;
      for (size_t i = 0; i < recomendados.size(); i++) {
        cout<<recomendados[i]<<" ";
      }
      cout<<endl;
    }
    else if(opcion == 2){
      cout<<endl;
      cout<<"Ingresar id pelicula"<<endl;
      cin>>id_pelicula;
      cout<<endl;
      float proyeccion = proyeccion_movies(d_values, d_row_ind, d_col_ind, d_ind_users, d_row_size, values, row_ind, col_ind, ind_users, row_size, id, 10);
      cout<<"Valor proyectado:  "<<proyeccion<<endl;
    }
    else{
      break;
    }

    cout<<endl;
    cout<<endl;



    /* code */
  }





  // float * buffer = new float[n_knn * 4];
  // std::ifstream infile("foobar.bin", ios::binary | ios::in);
  // infile.read((char *)buffer, sizeof(float) * n_knn * 4);
  //
  // int * buffer2 = new int[n_knn * 4];
  // std::ifstream infile2("foobar2.bin", ios::binary | ios::in);
  // infile2.read((char *)buffer2, sizeof(int) * n_knn * 4);
  //
  // for (size_t i = 0; i < n_knn * 4 ; i++) {
  //   if(buffer[i] < numeric_limits<float>::infinity())
  //     cout<<buffer2[i]<<" -> "<<buffer[i]<<endl;
  // }
  //








  return 0;
}
