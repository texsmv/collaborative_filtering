#include "hip/hip_runtime.h"
#include "sparse_operations.h"
#include <fstream>

#include <cstdlib>
#include <ctime>



int main(int argc, char const *argv[]) {



  float* values;
  int *row_ind, * col_ind;
  int * ind_users, *row_size;

  float* d_values;
  int *d_row_ind, * d_col_ind;
  int * d_ind_users, * d_row_size;

  float* distances = new float[n_users];
  float* d_distances;

  read_movieLens("ml-latest/ratings.csv", values, row_ind, col_ind, ind_users, row_size);

  d_values = cuda_array<float>(n_ratings);
  d_row_ind = cuda_array<int>(n_ratings);
  d_col_ind = cuda_array<int>(n_ratings);
  d_ind_users = cuda_array<int>(n_users);
  d_row_size = cuda_array<int>(n_users);

  d_distances = cuda_array<float>(n_users);

  cuda_H2D<float>(values, d_values, n_ratings);
  cuda_H2D<int>(row_ind, d_row_ind, n_ratings);
  cuda_H2D<int>(col_ind, d_col_ind, n_ratings);
  cuda_H2D<int>(ind_users, d_ind_users, n_users);
  cuda_H2D<int>(row_size, d_row_size, n_users);

  int block_size = 256;
  dim3 block =  dim3(block_size, 1, 1);
  dim3 grid =  dim3(ceil(n_users / block_size), 1);


  one2all<<<grid, block>>>(d_values, d_row_ind, d_col_ind, d_ind_users, d_row_size, d_distances, 3);

  CHECK(hipDeviceSynchronize());
  cuda_D2H<float>(d_distances, distances, n_users);



  // d_row_ind = <int> cuda_array(n_ratings);

  // for (size_t i = 0; i < 10; i++) {
  //   cout<<values[i]<<" --> "<<row_ind[i]<<" "<<col_ind[i]<<endl;
  // }
  //
  // for (size_t i = 0; i < 4; i++) {
  //   cout<<"Ind: "<<ind_users[i]<<" stride: "<<row_size[i]<<endl;
  // }
  // cout<<"Ind: "<<ind_users[n_users - 1]<<" stride: "<<row_size[n_users - 1]<<endl;
  //
  //
  // float* r = float_pointer(values, ind_users, 1);
  //
  // for (size_t i = 0; i < row_size[1]; i++) {
  //   cout<<r[i]<<" - ";
  // }
  // cout<<endl;
  //


  // float* distances2 = new float[n_users];
  //
  // start4 = clock();
  //
  //
  // float v;
  // for (size_t i = 3; i < 4; i++) {
  //   float* r1 = float_pointer(values, ind_users, i);
  //   int* c1 = int_pointer(col_ind, ind_users, i);
  //   // if(i % 1000 == 0)
  //   // cout<<i<<endl;
  //   for (size_t j = 0; j < n_users; j++) {
  //     float* r2 = float_pointer(values, ind_users, j);
  //     int* c2 = int_pointer(col_ind, ind_users, j);
  //     v = euclidean(r1, c1, row_size[i], r2, c2, row_size[j]);
  //     distances2[j] = v;
  //     // if(v != 0)
  //     //   cout<<v<<endl;
  //
  //   }
  // }
  //
  // stop4 = clock();
  // elapsed4 = (double)(stop4 - start4) * 1000.0 / CLOCKS_PER_SEC;
  // cout<<"Tiempo proyeccion:  "<<elapsed4<<"ms"<<endl;
  //

  // for (size_t i = 0; i < 100; i++) {
  //   cout<<distances2[i]<< " - "<<distances[i]<<endl;
  // }

  float *dists;
  int *ids;
  clock_t start4 = clock();
  n_knns(dists, ids, 0, 1000, d_values, d_row_ind, d_col_ind, d_ind_users, d_row_size);
  clock_t stop4 = clock();
  double elapsed4 = (double)(stop4 - start4) * 1000.0 / CLOCKS_PER_SEC;
  cout<<"Tiempo proyeccion:  "<<elapsed4<<"ms"<<endl;
  // knn(distances, ids, dists, 0, n_users);
  // cout<<"Mas cercanos"<<endl;
  // for (size_t i = 0; i < n_knn * 4 ; i++) {
  //   if(dists[i] < numeric_limits<float>::infinity())
  //     cout<<ids[i]<<" -> "<<dists[i]<<endl;
  // }

  std::ofstream ofile("foobar.bin", std::ios::binary);
  ofile.write((char*)dists, sizeof(float) * n_knn * 4);

  std::ofstream ofile2("foobar2.bin", std::ios::binary);
  ofile2.write((char*)ids, sizeof(int) * n_knn * 4);



  // float * buffer = new float[n_knn * 4];
  // std::ifstream infile("foobar.bin", ios::binary | ios::in);
  // infile.read((char *)buffer, sizeof(float) * n_knn * 4);
  //
  // int * buffer2 = new int[n_knn * 4];
  // std::ifstream infile2("foobar2.bin", ios::binary | ios::in);
  // infile2.read((char *)buffer2, sizeof(int) * n_knn * 4);
  //
  // for (size_t i = 0; i < n_knn * 4 ; i++) {
  //   if(buffer[i] < numeric_limits<float>::infinity())
  //     cout<<buffer2[i]<<" -> "<<buffer[i]<<endl;
  // }
  //








  return 0;
}
