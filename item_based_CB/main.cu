#include "hip/hip_runtime.h"
#include "scripts.h"
#include "cud_defs.h"
#include "cud_sparse_oper.h"
#include "recomender.h"


int main(int argc, char const *argv[]) {
  int n_ratings, n_users, n_movies, n_ids_movies;
  int n_ratings_20, n_users_20, n_ratings_27, n_users_27, n_movies_27;

  n_ratings_27 = 27753444;
  n_users_27 = 283228;
  n_movies_27 = 53889;


  n_ratings_20 = 20000263;
  n_users_20 = 138493;

  n_ratings = n_ratings_27;
  n_users = n_users_27;
  n_movies = n_movies_27;
  n_ids_movies = 193887;
  // n_ratings
  // n_of_users("../databases/ml-20m/ratings.csv", n_ratings, n_users, true);
  // cout<<n_ratings<<" "<<n_users<<endl;
  float* values;
  int *row_ind, * col_ind;
  int * ind_users, *row_size;
  float* maxs,*mins, *averages;
  int* pos_movies;

  float* item_values;
  int *item_row_ind, * item_col_ind;
  int * ind_items, *item_row_size;



  float* d_values;
  int *d_row_ind, * d_col_ind;
  int * d_ind_users, *d_row_size;
  float* d_maxs,*d_mins, *d_averages;

  float* d_item_values;
  int *d_item_row_ind, * d_item_col_ind;
  int * d_ind_items, *d_item_row_size;

  d_values = cuda_array<float>(n_ratings);
  d_row_ind = cuda_array<int>(n_ratings);
  d_col_ind = cuda_array<int>(n_ratings);
  d_ind_users = cuda_array<int>(n_users);
  d_row_size = cuda_array<int>(n_users);
  d_maxs = cuda_array<float>(n_users);
  d_mins = cuda_array<float>(n_users);
  d_averages = cuda_array<float>(n_users);


  d_item_values = cuda_array<float>(n_ratings);
  d_item_row_ind = cuda_array<int>(n_ratings);
  d_item_col_ind = cuda_array<int>(n_ratings);
  d_ind_items = cuda_array<int>(n_movies);
  d_item_row_size = cuda_array<int>(n_movies);


  string path = "../databases/ml-latest/ratings.csv";

  read_ML_ratings( path, n_ratings,  n_users, true  , values,row_ind, col_ind, ind_users, row_size,"27");
  read_ML_ratings_items(path, n_ratings, n_users, n_movies, n_ids_movies, true,  item_values,  item_row_ind,  item_col_ind,  ind_items, item_row_size, "27", pos_movies);
  average_per_user(values,ind_users, row_size,maxs,mins,averages,n_users);

  cuda_H2D<float>(values, d_values, n_ratings);
  cuda_H2D<int>(row_ind, d_row_ind, n_ratings);
  cuda_H2D<int>(col_ind, d_col_ind, n_ratings);
  cuda_H2D<int>(ind_users, d_ind_users, n_users);
  cuda_H2D<int>(row_size, d_row_size, n_users);
  cuda_H2D<float>(maxs, d_maxs, n_users);
  cuda_H2D<float>(mins, d_mins, n_users);
  cuda_H2D<float>(averages, d_averages, n_users);

  cuda_H2D<float>(item_values, d_item_values, n_ratings);
  cuda_H2D<int>(item_row_ind, d_item_row_ind, n_ratings);
  cuda_H2D<int>(item_col_ind, d_item_col_ind, n_ratings);
  cuda_H2D<int>(ind_items, d_ind_items, n_movies);
  cuda_H2D<int>(item_row_size, d_item_row_size, n_movies);

  float* similarity_matrix;

  float* posicion_sm = new float(0);
  if(!fexists("binary_files/posicion_sm")){
    cout<<"writing sm pos"<<endl;
    write_array<float>(posicion_sm, 1, "binary_files/posicion_sm");
  }
  else{
    read_array<float>(posicion_sm, 1, "binary_files/posicion_sm");
    cout<<"reading sm pos"<<endl;
  }
  cout<<"Posicion actual: "<<*posicion_sm<<endl;


  get_similarity_matrix(n_ratings, n_users, n_movies, d_item_values, d_item_row_ind, d_item_col_ind, d_ind_items, d_item_row_size, d_averages, similarity_matrix, posicion_sm);

  for (size_t i = 0; i < 4; i++) {
    for (size_t j = 0;  j< 4; j++) {
      /* code */
      cout<<get_tm2(i, j, similarity_matrix)<<" ";
    }
    cout<<endl;
  }

    /* code */
  float* r1 = float_pointer(values, ind_users, 3);
  int* c1 = int_pointer(col_ind, ind_users, 3);
  cout<<predecir(similarity_matrix, maxs, mins, r1, c1, row_size[3], pos_movies[169], 3, pos_movies)<<endl;
  cout<<predecir(similarity_matrix, maxs, mins, r1, c1, row_size[3], pos_movies[339], 3, pos_movies)<<endl;
  cout<<predecir(similarity_matrix, maxs, mins, r1, c1, row_size[3], pos_movies[349], 3, pos_movies)<<endl;
  cout<<predecir(similarity_matrix, maxs, mins, r1, c1, row_size[3], pos_movies[296], 3, pos_movies)<<endl;

  // for (size_t i = 0; i < 100 ; i++) {
  //   /* code */
  //   cout<<pos_movies[i]<<endl;
  //   cout<<predecir(similarity_matrix, maxs, mins, r1, c1, row_size[3], pos_movies[i], 3, pos_movies)<<endl;
  // }

  // cout<<get_tm(0, 0, similarity_matrix)<<endl;
  // cout<<get_tm(0, 1, similarity_matrix)<<endl;
  // cout<<get_tm(0, 2, similarity_matrix)<<endl;
  // cout<<get_tm(123, 123, similarity_matrix)<<endl;


  // int tam = int(((double)n_movies + 1) * (double)n_movies / 2);
  // similarity_matrix = new float[tam];
  // for (size_t i = 0; i < tam; i++) {
  //   similarity_matrix[i] = 1;
  // }
  // write_array<float>(similarity_matrix, tam, "prueba");



  // float block_size = 256;
  // dim3 block =  dim3(block_size, 1, 1);
  // dim3 grid =  dim3(ceil(n_movies / block_size), 1);
  //
  // float* distances;
  // distances = new float[n_movies];
  // float* d_distances = cuda_array<float>(n_movies);
  // int pos_movie = 0;
  // one2all_adjusted_cosine<<<grid, block>>>(d_averages, d_item_values, d_item_row_ind, d_item_col_ind, d_ind_items, d_item_row_size, d_distances, pos_movie, n_movies);
  // CHECK(hipDeviceSynchronize());
  // cuda_D2H<float>(d_distances, distances, n_movies);
  // CHECK(hipDeviceSynchronize());
  //
  // for (size_t i = 0; i < 300; i++) {
  //   cout<<distances[i]<<endl;
  // }

  // int N = 5;
  // int tam = (N + 1) * N / 2;
  // cout<<tam<<endl;
  // float t[15] = {1,2,3,4,5,6,7,8,9,10,11,12,13,14,15};
  // float* matriz = &t[0];
  //
  // set_tm(4,2,100, matriz);
  // cout<<get_tm(0,0, matriz)<<endl;
  // cout<<get_tm(1,4, matriz)<<endl;
  // cout<<get_tm(3,4, matriz)<<endl;
  // cout<<get_tm(4,2, matriz)<<endl;


  // for (size_t i = 0; i < n_users; i++) {
  //   cout<<maxs[i]<<endl;
  //   cout<<mins[i]<<endl;
  //   cout<<averages[i]<<endl;
  // }

  // for (size_t i = 0; i < 40; i++) {
  //   cout<<item_values[i]<<endl;
  //   cout<<item_row_ind[i]<<endl;
  //   cout<<item_col_ind[i]<<endl;
  //   cout<<ind_items[i]<<endl;
  //   cout<<item_row_size[i]<<endl;
  //   cout<<endl;
  // }


  return 0;
}
