#include "hip/hip_runtime.h"
#include "scripts.h"
#include "cud_defs.h"
#include "cud_sparse_oper.h"
#include "recomender.h"


int main(int argc, char const *argv[]) {
  // float x;
  // float y;
  // y = modf(3.0, &x);
  // cout<<"x: "<<x<<endl;
  // cout<<"y: "<<y<<endl;

  int n_ratings, n_users, n_movies, n_ids_movies;
  int n_ratings_20, n_users_20, n_ratings_27, n_users_27, n_movies_27;

  n_ratings_27 = 27753444;
  n_users_27 = 283228;
  n_movies_27 = 53889;


  n_ratings_20 = 20000263;
  n_users_20 = 138493;

  n_ratings = n_ratings_27;
  n_users = n_users_27;
  n_movies = n_movies_27;
  n_ids_movies = 193887;
  // n_ratings
  // n_of_users("../databases/ml-20m/ratings.csv", n_ratings, n_users, true);
  // cout<<n_ratings<<" "<<n_users<<endl;
  float* values;
  int *row_ind, * col_ind;
  int * ind_users, *row_size;
  int* pos_movies;

  float* item_values;
  int *item_row_ind, * item_col_ind;
  int * ind_items, *item_row_size;



  float* d_values;
  int *d_row_ind, * d_col_ind;
  int * d_ind_users, *d_row_size;

  float* d_item_values;
  int *d_item_row_ind, * d_item_col_ind;
  int * d_ind_items, *d_item_row_size;

  d_values = cuda_array<float>(n_ratings);
  d_row_ind = cuda_array<int>(n_ratings);
  d_col_ind = cuda_array<int>(n_ratings);
  d_ind_users = cuda_array<int>(n_users);
  d_row_size = cuda_array<int>(n_users);


  d_item_values = cuda_array<float>(n_ratings);
  d_item_row_ind = cuda_array<int>(n_ratings);
  d_item_col_ind = cuda_array<int>(n_ratings);
  d_ind_items = cuda_array<int>(n_movies);
  d_item_row_size = cuda_array<int>(n_movies);


  string path = "../databases/ml-latest/ratings.csv";

  read_ML_ratings( path, n_ratings,  n_users, true  , values,row_ind, col_ind, ind_users, row_size,"27");
  read_ML_ratings_items(path, n_ratings, n_users, n_movies, n_ids_movies, true,  item_values,  item_row_ind,  item_col_ind,  ind_items, item_row_size, "27", pos_movies);

  cuda_H2D<float>(values, d_values, n_ratings);
  cuda_H2D<int>(row_ind, d_row_ind, n_ratings);
  cuda_H2D<int>(col_ind, d_col_ind, n_ratings);
  cuda_H2D<int>(ind_users, d_ind_users, n_users);
  cuda_H2D<int>(row_size, d_row_size, n_users);

  cuda_H2D<float>(item_values, d_item_values, n_ratings);
  cuda_H2D<int>(item_row_ind, d_item_row_ind, n_ratings);
  cuda_H2D<int>(item_col_ind, d_item_col_ind, n_ratings);
  cuda_H2D<int>(ind_items, d_ind_items, n_movies);
  cuda_H2D<int>(item_row_size, d_item_row_size, n_movies);

  float* similarity_matrix;

  float* posicion_sm = new float(0);
  if(!fexists("binary_files/posicion_sm")){
    cout<<"writing sm pos"<<endl;
    write_array<float>(posicion_sm, 1, "binary_files/posicion_sm");
  }
  else{
    read_array<float>(posicion_sm, 1, "binary_files/posicion_sm");
    cout<<"reading sm pos"<<endl;
  }
  cout<<"Posicion actual: "<<*posicion_sm<<endl;


  get_similarity_matrix(n_ratings, n_users, n_movies, d_item_values, d_item_row_ind, d_item_col_ind, d_ind_items, d_item_row_size, similarity_matrix, posicion_sm);

  // float* r1 = float_pointer(values, ind_users, 0);
  // int* c1 = int_pointer(col_ind, ind_users, 0);
  // cout<<predecir(similarity_matrix, r1, c1, row_size[3], pos_movies[169], 3, pos_movies)<<endl;
  // cout<<predecir(similarity_matrix, r1, c1, row_size[3], pos_movies[339], 3, pos_movies)<<endl;
  // cout<<predecir(similarity_matrix, r1, c1, row_size[3], pos_movies[349], 3, pos_movies)<<endl;
  // cout<<predecir(similarity_matrix, r1, c1, row_size[3], pos_movies[296], 3, pos_movies)<<endl;
  // int poss = 0;
  // while (true) {
  //   cin>>poss;
  //   cout<<predecir(similarity_matrix, r1, c1, row_size[0], pos_movies[poss], poss, pos_movies)<<endl;
  // }

  // for (size_t i = 0; i < 20; i++) {
  //   for (size_t j = 0; j < 20; j++) {
  //     cout<<get_tm2(i, j, similarity_matrix)<<" ";
  //   }
  //   cout<<endl;
  // }


  // cout<<get_tm(0, 0, similarity_matrix)<<endl;
  // cout<<get_tm(0, 1, similarity_matrix)<<endl;
  // cout<<get_tm(0, 2, similarity_matrix)<<endl;
  // cout<<get_tm(123, 123, similarity_matrix)<<endl;


  // int tam = int(((double)n_movies + 1) * (double)n_movies / 2);
  // similarity_matrix = new float[tam];
  // for (size_t i = 0; i < tam; i++) {
  //   similarity_matrix[i] = 1;
  // }
  // write_array<float>(similarity_matrix, tam, "prueba");



  // float block_size = 256;
  // dim3 block =  dim3(block_size, 1, 1);
  // dim3 grid =  dim3(ceil(n_movies / block_size), 1);
  //
  // float* distances;
  // distances = new float[n_movies];
  // float* d_distances = cuda_array<float>(n_movies);
  // int pos_movie = 0;
  // one2all_desviacion<<<grid, block>>>(d_item_values, d_item_row_ind, d_item_col_ind, d_ind_items, d_item_row_size, d_distances, pos_movie, n_movies);
  // CHECK(hipDeviceSynchronize());
  // cuda_D2H<float>(d_distances, distances, n_movies);
  // CHECK(hipDeviceSynchronize());
  //
  // for (size_t i = 0; i < 100; i++) {
  //   // if(distances[i] < 0)
  //     cout<<"i: "<<i<<" "<<distances[i]<<endl;
  // }

  // int N = 5;
  // int tam = (N + 1) * N / 2;
  // cout<<tam<<endl;
  // float t[15] = {1,2,3,4,5,6,7,8,9,10,11,12,13,14,15};
  // float* matriz = &t[0];
  //
  // set_tm2(4,2,100, matriz);
  // set_tm2(4, 0, -200, matriz);
  //
  // cout<<get_tm(0,0, matriz)<<endl;
  // cout<<get_tm(1,4, matriz)<<endl;
  // cout<<get_tm(3,4, matriz)<<endl;
  // cout<<get_tm(4,2, matriz)<<endl;
  // cout<<endl;
  // cout<<get_tm2(0,0, matriz)<<endl;
  // cout<<get_tm2(1,4, matriz)<<endl;
  // cout<<get_tm2(3,4, matriz)<<endl;
  // cout<<get_tm2(4,3, matriz)<<endl;
  // cout<<get_tm2(4,2, matriz)<<endl;
  // cout<<get_tm2(0,4, matriz)<<endl;
  // cout<<get_tm2(4,0, matriz)<<endl;


  // for (size_t i = 0; i < n_users; i++) {
  //   cout<<maxs[i]<<endl;
  //   cout<<mins[i]<<endl;
  //   cout<<averages[i]<<endl;
  // }

  // for (size_t i = 0; i < 40; i++) {
  //   cout<<item_values[i]<<endl;
  //   cout<<item_row_ind[i]<<endl;
  //   cout<<item_col_ind[i]<<endl;
  //   cout<<ind_items[i]<<endl;
  //   cout<<item_row_size[i]<<endl;
  //   cout<<endl;
  // }


  return 0;
}
